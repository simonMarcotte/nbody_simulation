#include "hip/hip_runtime.h"
#include "body.h"
#include "simulation.h"
#include <GLFW/glfw3.h>
#include <vector>
#include <random>
#include <cmath>
#include <cstdio>
#include <chrono>
#include <thread>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#define SCREEN_WIDTH 800
#define SCREEN_HEIGHT 600

int main() {
    // Initialize GLFW.
    if (!glfwInit()) return -1;
    GLFWwindow* window = glfwCreateWindow(SCREEN_WIDTH, SCREEN_HEIGHT, "HIP Accelerated Galaxy Collision", NULL, NULL);
    if (!window) { 
        glfwTerminate(); 
        return EXIT_FAILURE; 
    }
    glfwMakeContextCurrent(window);

    // Set up orthographic projection.
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(-1, 1, -1, 1, -1, 1);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    // Enable blending and anti-aliasing
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
    glEnable(GL_POINT_SMOOTH);
    glHint(GL_POINT_SMOOTH_HINT, GL_NICEST);
    glPointSize(4.0f);

    // Initialize bodies (example: two galaxies with different colors).
    std::vector<nbody::Body> bodies;
    const float denseMass = 1000.0f;
    const float orbiterMass = 1.0f;
    const int numOrbiters = 10000;
    const float G = 1.0f;
    float timeScale = 0.005f;
    float dt = 0.001f * timeScale;
    float eps = 1e-5f;

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> distAngle(0.0f, 2.0f * M_PI);
    std::uniform_real_distribution<float> distUniform(0.0f, 1.0f);

    // ----- Left Galaxy -----
    nbody::Body leftCentral;
    leftCentral.pos[0] = -0.5f;
    leftCentral.pos[1] = 0.0f;
    leftCentral.vel[0] = 0.05f;
    leftCentral.vel[1] = 0.0f;
    leftCentral.mass   = denseMass;
    leftCentral.galaxy = 0;
    bodies.push_back(leftCentral);

    float leftGroupRadius = 0.2f;
    for (int i = 0; i < numOrbiters; i++) {
        nbody::Body b;
        float angle = distAngle(gen);
        float r = leftGroupRadius * std::sqrt(distUniform(gen));
        b.pos[0] = leftCentral.pos[0] + r * std::cos(angle);
        b.pos[1] = leftCentral.pos[1] + r * std::sin(angle);
        b.mass   = orbiterMass;
        float speed = std::sqrt(G * leftCentral.mass / (r + 0.001f));
        b.vel[0] = leftCentral.vel[0] - speed * std::sin(angle);
        b.vel[1] = leftCentral.vel[1] + speed * std::cos(angle);
        b.galaxy = 0;
        bodies.push_back(b);
    }

    // ----- Right Galaxy -----
    nbody::Body rightCentral;
    rightCentral.pos[0] = 0.5f;
    rightCentral.pos[1] = 0.0f;
    rightCentral.vel[0] = -0.05f;
    rightCentral.vel[1] = 0.0f;
    rightCentral.mass   = denseMass;
    rightCentral.galaxy = 1;
    bodies.push_back(rightCentral);

    float rightGroupRadius = 0.2f;
    for (int i = 0; i < numOrbiters; i++) {
        nbody::Body b;
        float angle = distAngle(gen);
        float r = rightGroupRadius * std::sqrt(distUniform(gen));
        b.pos[0] = rightCentral.pos[0] + r * std::cos(angle);
        b.pos[1] = rightCentral.pos[1] + r * std::sin(angle);
        b.mass   = orbiterMass;
        float speed = std::sqrt(G * rightCentral.mass / (r + 0.001f));
        b.vel[0] = rightCentral.vel[0] - speed * std::sin(angle);
        b.vel[1] = rightCentral.vel[1] + speed * std::cos(angle);
        b.galaxy = 1;
        bodies.push_back(b);
    }

    // Click to start animation
    bool simulationStarted = false;
    while (!simulationStarted && !glfwWindowShouldClose(window)) {
        glfwPollEvents();
        if (glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_LEFT) == GLFW_PRESS) {
            simulationStarted = true;
        }
        std::this_thread::sleep_for(std::chrono::milliseconds(10));
    }

    // Simulation loop
    while (!glfwWindowShouldClose(window)) {
        simulationUpdate(bodies, dt, G, eps); // HIP implementation

        // Clear background with a dark color.
        glClearColor(0.05f, 0.05f, 0.1f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);
        glLoadIdentity();

        // Render bodies.
        glBegin(GL_POINTS);
        for (const auto &b : bodies) {
            if (b.galaxy == 0) {
                glColor4f(1.0f, 0.5f, 0.0f, 0.8f);  // Orange
            } else if (b.galaxy == 1) {
                glColor4f(0.0f, 0.0f, 1.0f, 0.8f);  // Blue
            } else {
                glColor4f(1.0f, 1.0f, 1.0f, 0.8f);
            }
            glVertex2f(b.pos[0], b.pos[1]);
        }
        glEnd();

        glfwSwapBuffers(window);
        glfwPollEvents();
        std::this_thread::sleep_for(std::chrono::milliseconds(1));
    }

    glfwDestroyWindow(window);
    glfwTerminate();
    return EXIT_SUCCESS;
}
