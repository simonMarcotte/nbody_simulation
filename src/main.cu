#include "hip/hip_runtime.h"
#include "body.h"
#include "simulation.h"
#include <GLFW/glfw3.h>
#include <vector>
#include <random>
#include <cmath>
#include <cstdio>
#include <chrono>
#include <thread>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#define SCREEN_WIDTH 800
#define SCREEN_HEIGHT 600

int main() {
    // Initialize GLFW.
    if (!glfwInit()) return -1;
    GLFWwindow* window = glfwCreateWindow(SCREEN_WIDTH, SCREEN_HEIGHT, "HIP Accelerated Galaxy Collision", NULL, NULL);
    if (!window) { glfwTerminate(); return -1; }
    glfwMakeContextCurrent(window);

    // Set up orthographic projection.
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(-1, 1, -1, 1, -1, 1);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glPointSize(2.0f);

    // Initialize bodies (example: two galaxies with different colors).
    std::vector<nbody::Body> bodies;
    const float denseMass = 1000.0f;
    const float orbiterMass = 1.0f;
    const int numOrbiters = 10000;
    const float G = 1.0f;
    float timeScale = 0.01f;
    float dt = 0.001f * timeScale;
    float eps = 1e-5f;


    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> distAngle(0.0f, 2.0f * M_PI);
    std::uniform_real_distribution<float> distUniform(0.0f, 1.0f);

    // ----- Left Galaxy -----
    // Central body for the left galaxy.
    nbody::Body leftCentral;
    leftCentral.pos[0] = -0.5f;
    leftCentral.pos[1] = 0.0f;
    leftCentral.vel[0] = 0.05f;  // x vel
    leftCentral.vel[1] = 0.0f;
    leftCentral.mass   = denseMass;
    leftCentral.galaxy = 0;
    bodies.push_back(leftCentral);

    float leftGroupRadius = 0.2f;
    for (int i = 0; i < numOrbiters; i++) {
        nbody::Body b;
        float angle = distAngle(gen);
        float r = leftGroupRadius * std::sqrt(distUniform(gen));
        b.pos[0] = leftCentral.pos[0] + r * std::cos(angle);
        b.pos[1] = leftCentral.pos[1] + r * std::sin(angle);
        b.mass   = orbiterMass;
        float speed = std::sqrt(G * leftCentral.mass / (r + 0.001f));
        // Set velocity perpendicular to the radius vector (for a circular orbit)
        b.vel[0] = leftCentral.vel[0] - speed * std::sin(angle);
        b.vel[1] = leftCentral.vel[1] + speed * std::cos(angle);
        b.galaxy = 0;  // Same galaxy ID.
        bodies.push_back(b);
    }

    // ----- Right Galaxy -----
    // Central body for the right galaxy.
    nbody::Body rightCentral;
    rightCentral.pos[0] = 0.5f;
    rightCentral.pos[1] = 0.0f;
    rightCentral.vel[0] = -0.05f;  // x vel
    rightCentral.vel[1] = 0.0f;
    rightCentral.mass   = denseMass;
    rightCentral.galaxy = 1;
    bodies.push_back(rightCentral);

    float rightGroupRadius = 0.2f;
    for (int i = 0; i < numOrbiters; i++) {
        nbody::Body b;
        float angle = distAngle(gen);
        float r = rightGroupRadius * std::sqrt(distUniform(gen));
        b.pos[0] = rightCentral.pos[0] + r * std::cos(angle);
        b.pos[1] = rightCentral.pos[1] + r * std::sin(angle);
        b.mass   = orbiterMass;
        float speed = std::sqrt(G * rightCentral.mass / (r + 0.001f));
        b.vel[0] = rightCentral.vel[0] - speed * std::sin(angle);
        b.vel[1] = rightCentral.vel[1] + speed * std::cos(angle);
        b.galaxy = 1;
        bodies.push_back(b);
    }

    std::printf("Total bodies: %lu\n", bodies.size());


    while (!glfwWindowShouldClose(window)) {

        simulationUpdate(bodies, dt, G, eps); // HIP implementation

        // Rendering
        glClear(GL_COLOR_BUFFER_BIT);
        glLoadIdentity();
        glBegin(GL_POINTS);
        for (const auto &b : bodies) {
            if (b.galaxy == 0) {
                glColor3f(1.0f, 0.5f, 0.0f);  // Orange
            } else if (b.galaxy == 1) {
                glColor3f(0.0f, 0.0f, 1.0f);  // Blue
            } else {
                glColor3f(1.0f, 1.0f, 1.0f);
            }
            glVertex2f(b.pos[0], b.pos[1]);
        }
        glEnd();
        glfwSwapBuffers(window);
        glfwPollEvents();

        std::this_thread::sleep_for(std::chrono::milliseconds(1));
    }

    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}