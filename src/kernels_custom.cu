#include "hip/hip_runtime.h"
#include "kernels_custom.h"
#include <math.h>

__global__ void updateBodiesKernel(float* pos, float* vel, const float* mass, int n, float dt, float G, float eps) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        // Each body has 2 components (x and y)
        float xi = pos[2 * i];
        float yi = pos[2 * i + 1];
        float ax = 0.0f;
        float ay = 0.0f;
        for (int j = 0; j < n; j++) {
            if (i == j) continue;
            float dx = pos[2 * j] - xi;
            float dy = pos[2 * j + 1] - yi;
            float distSq = dx * dx + dy * dy + eps;
            float r = sqrtf(distSq);
            float a = G * mass[j] / distSq;
            ax += a * dx / r;
            ay += a * dy / r;
        }
        // Update velocity and position
        vel[2 * i]     += ax * dt;
        vel[2 * i + 1] += ay * dt;
        pos[2 * i]     += vel[2 * i] * dt;
        pos[2 * i + 1] += vel[2 * i + 1] * dt;
    }
}
